#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#define N 23 * 1024
#define ThreadsPerBlock 256
#define BlocksPerGrid (N + ThreadsPerBlock - 1) / ThreadsPerBlock

__global__ void addKernel(const int *a, int *r)
{
	__shared__ int cache[ThreadsPerBlock];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int cacheIndex = threadIdx.x;

	// copy data to shared memoryfrom global memory
	cache[cacheIndex] = a[tid];
	__syncthreads();

	// add these data using reduce
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		if ((tid % (2 * i)) == 0)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
	}

	// copy the result of reduce to global memory
	if (cacheIndex == 0)
		r[blockIdx.x] = cache[cacheIndex];
}

int main()
{
	int *a, *result;
	int *dev_a, *dev_result;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory in CPU
	a = (int*)malloc(N * sizeof(int));
	result = (int*)malloc(BlocksPerGrid * sizeof(int));
	// allocate memory in GPU
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_result, BlocksPerGrid * sizeof(int));

	// generate data for a
	for (int i = 0; i < N; ++i)
		a[i] = 1;

	hipEventRecord(start, 0);
	// copy data from CPU to GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

	addKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_a, dev_result);

	// copy result from GPU to CPU
	hipMemcpy(result, dev_result, BlocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 1; i < BlocksPerGrid; ++i)
		result[0] += result[i];
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Is the result %d equal to %d?\n", result[0], N);
	printf("Time of computing is %.3lf", elapsedTime);

	// free
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_result);
	free(a);
	free(result);
}